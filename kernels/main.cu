#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ void dev_sweep(gpuCam* cam, char* x)
{
	*x = cam[0].name[2];
}

gpuCam* transform_cams(vector<cam> const& cam_vector)
{
	gpuCam* cameras = (gpuCam*)malloc(sizeof(gpuCam) * cam_vector.size());

	for (int i = 0; i < cam_vector.size(); i++)
	{
		cam camera = cam_vector.at(i);
		char* name;

		hipMalloc((void**)&name, camera.name.size());
		hipMemcpy(name, camera.name.data(), camera.name.size(), hipMemcpyHostToDevice);

		cameras[i].size = camera.size;
		cameras[i].width = camera.width;
		cameras[i].height = camera.height;
		cameras[i].name = name;
	}

	return cameras;
}

void gpu_sweeping_plane(cam const ref, std::vector<cam> const& cam_vector, int window)
{
	// 1 - Preprocess data and set GPU device
	hipSetDevice(0);

	gpuCam* gpuCams = transform_cams(cam_vector);
	gpuCam* dev_gpuCameras;
	char* dev_result;

	hipMalloc((void**)&dev_gpuCameras, sizeof(gpuCam) * cam_vector.size());
	hipMalloc((void**)&dev_result, sizeof(char));

	// 2 - Copy data to GPU
	hipMemcpy(dev_gpuCameras, gpuCams, sizeof(gpuCam) * cam_vector.size(), hipMemcpyHostToDevice);

	// 3 - Launch kernel and wait for it to finish
	const dim3 nbBlocks(1);
	const dim3 nbThreadsPerBlock(1);

	dev_sweep << <nbBlocks, nbThreadsPerBlock >> > (dev_gpuCameras, dev_result);
	hipDeviceSynchronize();
	
	// 4 - Extract result from GPU
	char res = 0;
	hipMemcpy(&res, dev_result, sizeof(char), hipMemcpyDeviceToHost);

	cout << res << endl;
	cout << cam_vector.at(0).name.at(2) << endl;
	
	// 5 - Free CPU and/or GPU memory
	for (int i = 0; i < cam_vector.size(); i++)
	{
		hipFree(&(dev_gpuCameras[i].name));
		hipFree(&(gpuCams[i].name));
	}

	hipFree(dev_gpuCameras);
	hipFree(gpuCams);

	hipFree(dev_result);
	
	// 6 - Build & return cost cube	(TODO)
}
