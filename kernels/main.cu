#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ void dev_sweep(gpuCam* cam, int ref_cam_index, float* result)
{
	result[2084] = (float)ref_cam_index + 168;
}

gpuCam* transform_cams(vector<cam> const& cam_vector)
{
	gpuCam* cameras = (gpuCam*)malloc(sizeof(gpuCam) * cam_vector.size());

	for (int i = 0; i < cam_vector.size(); i++)
	{
		cam camera = cam_vector.at(i);

		char* name;
		hipMalloc((void**)&name, camera.name.size());
		hipMemcpy(name, camera.name.data(), camera.name.size(), hipMemcpyHostToDevice);

		double* K;
		hipMalloc((void**)&K, camera.p.K.size() * sizeof(double));
		hipMemcpy(K, camera.p.K.data(), camera.p.K.size() * sizeof(double), hipMemcpyHostToDevice);

		double* R;
		hipMalloc((void**)&R, camera.p.R.size() * sizeof(double));
		hipMemcpy(R, camera.p.R.data(), camera.p.R.size() * sizeof(double), hipMemcpyHostToDevice);

		double* t;
		hipMalloc((void**)&t, camera.p.t.size() * sizeof(double));
		hipMemcpy(t, camera.p.t.data(), camera.p.t.size() * sizeof(double), hipMemcpyHostToDevice);

		double* K_inv;
		hipMalloc((void**)&K_inv, camera.p.K_inv.size() * sizeof(double));
		hipMemcpy(K_inv, camera.p.K_inv.data(), camera.p.K_inv.size() * sizeof(double), hipMemcpyHostToDevice);

		double* R_inv;
		hipMalloc((void**)&R_inv, camera.p.R_inv.size() * sizeof(double));
		hipMemcpy(R_inv, camera.p.R_inv.data(), camera.p.R_inv.size() * sizeof(double), hipMemcpyHostToDevice);

		double* t_inv;
		hipMalloc((void**)&t_inv, camera.p.t_inv.size() * sizeof(double));
		hipMemcpy(t_inv, camera.p.t_inv.data(), camera.p.t_inv.size() * sizeof(double), hipMemcpyHostToDevice);

		uint8_t* YChannelData;
		hipMalloc((void**)&YChannelData, camera.YUV[0].total() * camera.YUV[0].elemSize());
		hipMemcpy(YChannelData, camera.YUV[0].data, camera.YUV[0].total() * camera.YUV[0].elemSize(), hipMemcpyHostToDevice);

		cameras[i].size = camera.size;
		cameras[i].width = camera.width;
		cameras[i].height = camera.height;
		cameras[i].name = name;
		cameras[i].K = K;
		cameras[i].R = R;
		cameras[i].t = t;
		cameras[i].K_inv = K_inv;
		cameras[i].R_inv = R_inv;
		cameras[i].t_inv = t_inv;
		cameras[i].YChannelData = YChannelData;
	}

	return cameras;
}

void gpu_sweeping_plane(std::vector<cam> const& cam_vector, int ref_cam_index, int window)
{
	// 1 - Preprocess data and set GPU device
	hipSetDevice(0);

	gpuCam* gpuCams = transform_cams(cam_vector);
	gpuCam* dev_gpuCameras;
	float* dev_result;

	hipMalloc((void**)&dev_gpuCameras, sizeof(gpuCam) * cam_vector.size());

	int resSize = ZPlanes * cam_vector.at(0).width * cam_vector.at(0).height * sizeof(float);
	hipMalloc((void**)&dev_result, resSize);

	// 2 - Copy data to GPU
	hipMemcpy(dev_gpuCameras, gpuCams, sizeof(gpuCam) * cam_vector.size(), hipMemcpyHostToDevice);

	// 3 - Launch kernel and wait for it to finish
	const dim3 nbBlocks(1);
	const dim3 nbThreadsPerBlock(1);

	dev_sweep << <nbBlocks, nbThreadsPerBlock >> > (dev_gpuCameras, ref_cam_index, dev_result);
	hipDeviceSynchronize();

	// 4 - Extract result from GPU
	float* res = (float*)malloc(resSize);
	hipMemcpy(res, dev_result, resSize, hipMemcpyDeviceToHost);

	// 5 - Free CPU and/or GPU memory
	for (int i = 0; i < cam_vector.size(); i++)
	{
		hipFree(&(dev_gpuCameras[i].name));
		hipFree(&(gpuCams[i].name));
		hipFree(&(dev_gpuCameras[i].K));
		hipFree(&(gpuCams[i].K));
		hipFree(&(dev_gpuCameras[i].R));
		hipFree(&(gpuCams[i].R));
		hipFree(&(dev_gpuCameras[i].t));
		hipFree(&(gpuCams[i].t));
		hipFree(&(dev_gpuCameras[i].K_inv));
		hipFree(&(gpuCams[i].K_inv));
		hipFree(&(dev_gpuCameras[i].R_inv));
		hipFree(&(gpuCams[i].R_inv));
		hipFree(&(dev_gpuCameras[i].t_inv));
		hipFree(&(gpuCams[i].t_inv));
		hipFree(&(dev_gpuCameras[i].YChannelData));
		hipFree(&(gpuCams[i].YChannelData));
	}

	hipFree(dev_gpuCameras);
	hipFree(gpuCams);

	hipFree(dev_result);

	// 6 - Build & return cost cube	(TODO) + free its memory space

	free(res);
}